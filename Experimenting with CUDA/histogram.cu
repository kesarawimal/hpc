#include "hip/hip_runtime.h"
6CS005 - High Performance Computing
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>

/*****************************************************************************

 This version enables the number of blocks to be varied, as opposed to the 
 number of threads per block being varied in the previous version. This
 version has had a change to the main method to vary the number of blocks of
 threads. Every time it iterates it increases the number of blocks. The 
 results are very interesting and might not be what you expect. You can think 
 of it as being very similar to the car wash problem talked about in the 
 lecture. You might want to do some more formal testing where each experiment
 is performed multiple times and the means taken.  

 To compile:
    nvcc -o war_and_peace_05 war_and_peace_05.cu -lrt


  Dr K A Buckley, University of Wolverhampton, 2018

*****************************************************************************/

long int host_n_bytes, *dev_n_bytes;
int n_bins = 256;
unsigned char *host_data, *dev_data;
int *host_bins, *dev_bins;
int n_blocks;

/**
  Given two time structures calculate the difference between them. Result
  is returned through the third parameter. Function returns 0 on success.
*/

int time_difference(struct timespec *start, struct timespec *finish, 
  long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

/**
  Allocates memory for the book data and histogram and loads the book data
  from file. You need to have downloaded the file for this function to work.
*/

void load_data(char *file_name){
  hipError_t error;
  long bytes_read;
  struct stat stbuf;
 
  if(stat(file_name, &stbuf) == -1){
    fprintf(stderr, "Problem accessing file status\n");
    exit(0);
  }
  host_n_bytes = stbuf.st_size;

  host_data = (unsigned char *) malloc(sizeof(unsigned char) * host_n_bytes);
  host_bins = (int *) malloc(sizeof(int) * n_bins);
  
  FILE *f = fopen(file_name, "r");
  if(!f){
    fprintf(stderr, "Problem opening file\n");
    exit(1);        
  }

  bytes_read = fread(host_data, 1, host_n_bytes, f);
  if(bytes_read != host_n_bytes){
    fprintf(stderr, "Problem reading file. Not enough data read.\n");
    exit(1);        
  }
  fclose(f);  

  error = hipMalloc(&dev_data, host_n_bytes); 
  if(error){
    fprintf(stderr, "malloc of dev_data %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }

  error = hipMalloc(&dev_bins, sizeof(int) * n_bins); 
  if(error){
    fprintf(stderr, "malloc of dev_bins %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
  
  error = hipMalloc(&dev_n_bytes, sizeof(long int)); 
  if(error){
    fprintf(stderr, "malloc of n_bytes %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
  
  error = hipMemcpy(dev_data, host_data, host_n_bytes, 
    hipMemcpyHostToDevice);
  if(error){
    fprintf(stderr, "copy from host_data to dev_data %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
  
  error = hipMemcpy(dev_n_bytes, &host_n_bytes, sizeof(long int), 
    hipMemcpyHostToDevice);
  if(error){
    fprintf(stderr, "copy from host_n_bytes to dev_n_bytes %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
}

/**
  Iterates through the characters in the data incrementing the appropriate 
  histogram bins.
*/

__global__ void kernel_n_256(unsigned char *data, 
                             int *bins, long int *n_bytes){
  int i, j, k, stride, start;
  i = threadIdx.x;
  k = blockIdx.x;
  stride = gridDim.x;
  start = blockIdx.x;
  
  printf("thread %d,%d starting at %d using stride %d\n", i, k, start, stride);
  
  for(j=start;j<*n_bytes;j+=stride){
    if(data[j] == i){
      atomicAdd(&bins[i], 1);
    }
  }   
}

/**
  Launch threads. Each one will look for the particular character that it is
  associated with and adjust the required histogram bin.
*/

void compute_histogram(){
  hipError_t error;

  kernel_n_256 <<<n_blocks, 256>>>(dev_data, dev_bins, dev_n_bytes);
  error = hipGetLastError();

  if(error){
    fprintf(stderr, "Kernel launch returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
  error = hipDeviceSynchronize();
  if(error){
    fprintf(stderr, "Thread synchonize returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
}

/**
  Time the calculation of the histogram then output and check the results.
*/

void run() {
  int i;
  hipError_t error;
  struct timespec start, finish;   
  long long int time_elapsed;
 
  clock_gettime(CLOCK_MONOTONIC, &start);
  compute_histogram();
  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  
  error = hipMemcpy(host_bins, dev_bins, n_bins * sizeof(int), 
    hipMemcpyDeviceToHost);
  if(error){
    fprintf(stderr, "copy from dev_bins to host_bins %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }  
  printf("\nresults\n========\n");
  int count = 0;
  for(i=0;i<n_bins;i++){
    if(i>=32 && i<127) {
      printf("%3d: %c: %d\n", i, i, host_bins[i]);
    }
    count += host_bins[i];
  }

  printf("\n%d records were found in bins\n", count);
  if(count != host_n_bytes){
    printf("Results are incorrect\n");
    printf("%ld bytes from %ld are missing\n", 
      host_n_bytes - count, host_n_bytes);
  } else {
    printf("results are correct\n");
  }
  printf("run took %0.9lfs\n", (time_elapsed/1.0e9)); 
}

/**
  Sets all histogram bins to zero so that the program can repeatedly
  calculate the histogram to measure performance.
*/

void clear_results(){
  hipError_t error;
  error = hipMemset(dev_bins, 0, n_bins * sizeof(int));
  if(error){
    fprintf(stderr, "Mem set of dev_bins returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
}

/**
  Free up all memory that was dynamically allocated.
*/

void cleanup(){
  hipError_t error;
  
  error = hipFree(dev_data);
    if(error){
    fprintf(stderr, "Free dev_data returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }
  error = hipFree(dev_bins);
  if(error){
    fprintf(stderr, "Free dev_bins returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }  
  error = hipFree(dev_n_bytes);
  if(error){
    fprintf(stderr, "Free dev_n_bytes returned %d %s\n", error,
      hipGetErrorString(error));
    exit(1);
  }  
  free(host_data);
  free(host_bins);
}

int main(int argc, char * argv[]){
  int i;
  
  if(argc != 2){
    fprintf(stderr, "You need to specify the path/file name of the book\n");
    exit(1);
  }

  load_data(argv[1]);
  for(i=1;i<128;i++){
    n_blocks = i;
    clear_results();
    run();
  }
  
  cleanup();
}